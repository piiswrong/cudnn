#include <common.cuh>

hipblasStatus_t  cublasXasum(hipblasHandle_t handle, int n,
                            const float           *x, int incx, float  *result)
{
    return hipblasSasum(handle, n, x, incx, result);
}

hipblasStatus_t  cublasXasum(hipblasHandle_t handle, int n,
                            const double          *x, int incx, double *result)
{
    return hipblasDasum(handle, n, x, incx, result);
}					
							
hipblasStatus_t  cublasXnrm2(hipblasHandle_t handle, int n,
                            const float           *x, int incx, float  *result)
{
    return hipblasSnrm2(handle, n, x, incx, result);
}

hipblasStatus_t  cublasXnrm2(hipblasHandle_t handle, int n,
                            const double          *x, int incx, double *result)
{
    return hipblasDnrm2(handle, n, x, incx, result);
}
hipblasStatus_t cublasXaxpy(hipblasHandle_t handle, int n,
                                        const float           *alpha,
                                        const float           *x, int incx,
                                        float                 *y, int incy) 
{
    return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasXaxpy(hipblasHandle_t handle, int n,
                                        const double          *alpha,
                                        const double          *x, int incx,
                                        double                *y, int incy)
{
    return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasXgemm(hipblasHandle_t handle,
                                        hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k,
                                        const float *alpha, float *A, int lda,
                                        float *B, int ldb, const float *beta,
                                        float *C, int ldc)
{
    return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

hipblasStatus_t cublasXgemm(hipblasHandle_t handle,
                                        hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k,
                                        const double *alpha, double *A, int lda,
                                        double *B, int ldb, const double *beta,
                                        double *C, int ldc)
{
    return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}


