#include "hip/hip_runtime.h"
#include <common.cuh>
#include <DNN.cuh>
#include <DData.cuh>

#ifdef NVML
#include <nvml_old.h>
#endif
#include <sstream>

template<class T>
void fineTuneWithCheckpoint(DNN<T> *dnn, DData<T> *data, int ntotal, int ninterval, std::string path, int resuming) {
    for (int i = 0; i < ntotal; i += ninterval) {
        if (i + ninterval <= ntotal) 
            dnn->fineTune(data, ninterval);
        else 
            dnn->fineTune(data, ntotal - i);
        std::stringstream ss;
        ss << i+resuming;
        FILE *fout = fopen((path+"_"+ss.str()+".param").c_str(), "w");
        dnn->save(fout);
        fclose(fout);
    }
}

int main(int argc, char **argv) {
    std::string path = "/projects/grail/jxie/cudnn/log/";
    //feenableexcept(FE_DIVBYZERO | FE_INVALID | FE_OVERFLOW);
#ifdef USE_MPI 
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_world_rank);
#endif

    FILE *fin = NULL;
    char * exp_name = NULL;
    int resuming = -1;
    int devId = -1;
    bool grad_check = false;
    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-') {
            switch (argv[i][1]) {
            case 'r': resuming = atoi(argv[i+1]); ++i; break;
            case 'd': devId = atoi(argv[i+1]); i++; break;
            case 'c': grad_check = true; break;
            default: printf("Invalid command line argument \'%c\'!\n", argv[i][1]); exit(-1); break;
            }
        }else {
            exp_name = argv[i];
        }
    }
    if (exp_name != NULL) {
        flog = fopen((path+exp_name+".log").c_str(), "w");
        fin = fopen((path+exp_name+".hyper").c_str(), "r");
        printf("Using configuration file %s\n", (path+exp_name+".hyper").c_str());
        if (fin == NULL) exit(-1);
    }

#ifndef DISABLE_GPU
#ifdef NVML
    if (devId == -1) {
        nvmlReturn_t ret;
        unsigned int deviceCount;

        if ((ret = nvmlInit()) != NVML_SUCCESS)
        {
            printf("Could not init NVML: %s\n", nvmlErrorString(ret));
            return 1;
        }

        if ((ret = nvmlDeviceGetCount(&deviceCount)) != NVML_SUCCESS)
        {
            printf("Could not get device count: %s\n", nvmlErrorString(ret));
            nvmlShutdown();
            return 1;
        }
        printf("Device count: %d\n", deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            nvmlDevice_t device;
            if ((ret = nvmlDeviceGetHandleByIndex(i, &device)) != NVML_SUCCESS)
            {
                printf("Skip %d, can not get index\n", i);
                continue;
            }
            nvmlUtilization_t util;
            if ((ret = nvmlDeviceGetUtilizationRates(device, &util)) != NVML_SUCCESS)
            {
                printf("Can not get util rate on %d\n", i);
            }else {
                if (util.gpu < 5) 
                {
                    devId = i;
                    break;
                }
            }
        }
    }

    if (devId != -1)
    {
        printf("Selecting device %d\n", devId);
        CUDA_CALL(hipSetDevice(devId));
    }else {
        printf("Can not find idle device\n");
        exit(-1);
    }
#else
    devId = 0;
#endif
#endif


    hipblasHandle_t handle = 0; 
    CUBLAS_CALL(hipblasCreate(&handle));

    int num_layers = 2;
    int hidden_dim = 1024;
    //int input_dim = 351, output_dim = 150;
    int input_dim = 1568, output_dim = 256;
    //int input_dim = 28*28, output_dim = 10;
    //int input_dim = 32, output_dim = 32;
    char unit[255];
    strcpy(unit, "ReLU");
    float pt_epochs = 0.0;
    int bp_epochs = 100;
    DHyperParams _bp_hyper_params, _pt_hyper_params;
    _pt_hyper_params.idrop_out = false;
    _pt_hyper_params.idrop_rate = 0.2;
    _pt_hyper_params.hdrop_out = false;
    _pt_hyper_params.weight_decay = false;
    _pt_hyper_params.decay_rate = 0.00;
    _pt_hyper_params.momentum = 0.90;
    _pt_hyper_params.learning_rate = 0.01;

    _bp_hyper_params.check_interval = 128;
    _bp_hyper_params.learning_rate = 0.5;
    _bp_hyper_params.idrop_out = false;
    _bp_hyper_params.idrop_rate = 0.2;
    _bp_hyper_params.hdrop_out = true;
    _bp_hyper_params.hdrop_rate= 0.5;
    _bp_hyper_params.momentum = 0.5;
    _bp_hyper_params.max_momentum = 0.90;
    _bp_hyper_params.step_momentum = 0.04;
    _bp_hyper_params.weight_decay = false;
    _bp_hyper_params.decay_rate = 0.01;

#ifdef ADMM
    _bp_hyper_params.decay_rate = 0.001;
#endif

    if (fin != NULL) {
        READ_PARAM(num_layers);
        READ_PARAM(hidden_dim);
        fscanf(fin, "neuron=%s\n", unit);
        READ_PARAM(pt_epochs);
        READ_PARAM(bp_epochs);
        _pt_hyper_params.load(fin);
        _bp_hyper_params.load(fin);
    }


    int *layer_dims = new int[num_layers+1];
    layer_dims[0] = input_dim;
    layer_dims[num_layers] = output_dim;
    for (int i = 1; i < num_layers; i++) layer_dims[i] = hidden_dim;

    DNeuron<float> **neuron = new DNeuron<float>*[num_layers];
    for (int i = 0; i < num_layers-1; i++) {
        std::string str_unit(unit);
        if (str_unit == "Logistic") {
            neuron[i] = new DLogisticNeuron<float>(handle);
        }else if (str_unit == "Oddroot") {
            printf("using oddroot\n");
            neuron[i] = new DOddrootNeuron<float>(handle);
        }else if (str_unit == "ReLU") {
            neuron[i] = new DReLUNeuron<float>(handle);
        }else if (str_unit == "Linear") {
            neuron[i] = new DNeuron<float>(handle);
        }else {
            printf("ERROR: \"%s\" is not a supported neuron type\n", unit);
            exit(-1);
        }
    }
#ifdef ADMM
    DParallelMnistData<float> *data = new DParallelMnistData<float>("../data", mpi_world_size, mpi_world_rank, _bp_hyper_params.batch_size, dnn->handle());
    data->set_devId(devId);
    dnn->admmFineTune(data, 500);
#elif defined(DOWN_POUR_SGD)
    DParallelMnistData<float> *data = new DParallelMnistData<float>("../data", mpi_world_size - sgd_num_param_server, mpi_world_rank - sgd_num_param_server, _bp_hyper_params.batch_size, dnn->handle());
    data->set_devId(devId);
    dnn->fineTune(data, 500);

#else
    //DMnistData<float> *data = new DMnistData<float>("../data/", DData<float>::Train, 50000, false, dnn->handle());
    //DData<float> *data = new DDummyData<float>(input_dim, 1, handle);
    //DTimitData<float> *data = new DTimitData<float>("/scratch/jxie/", 10000, false, dnn->handle());
    DData<float> *data = new DPatchData<float>("/projects/grail/jxie/paris/", input_dim, 10000, false, handle);
#ifndef DISABLE_GPU
    data->set_devId(devId);
#endif

    //neuron[num_layers-1] = new DSoftmaxNeuron<float>(_bp_hyper_params.batch_size, handle);
    //neuron[num_layers-1] = new DGMMNeuron<float>(&_bp_hyper_params, 256, output_dim, 0.1, handle);
    DvMFNeuron<float> *last_neuron = new DvMFNeuron<float>(&_bp_hyper_params, 32, output_dim, 0.2, handle);
    last_neuron->init(data);
    neuron[num_layers-1] = last_neuron;
    
    DNN<float> *dnn = new DNN<float>(num_layers, layer_dims, neuron, &_pt_hyper_params, &_bp_hyper_params, handle);

    if (grad_check) {
        return !dnn->createGradCheck(data);
    }
    if (resuming == -1 && pt_epochs > 0) dnn->pretrain(data, pt_epochs);
    if (resuming != -1) {
        printf("Resuming from %d-th epoch.\n", resuming);
        //std::stringstream ss;
        //ss << resuming - 10;
        fin = fopen((path+exp_name+".param").c_str(), "r");
        if (fin == 0) {
            printf("Error loading: cannot find file %s!\n", (path+exp_name+".param").c_str());
            exit(-1);
        }
        dnn->layers()[0]->weight()->samplePrint();
        dnn->load(fin);
        dnn->layers()[0]->weight()->samplePrint();
        fclose(fin);
        _bp_hyper_params.learning_rate *= std::pow(_bp_hyper_params.learning_rate_decay, resuming);
    }else 
        resuming = 0;
    if (exp_name != NULL)
        fineTuneWithCheckpoint(dnn, data, bp_epochs, 10, path+exp_name, resuming);
    else 
        dnn->fineTune(data, bp_epochs);

#endif
    if (exp_name != NULL) {
        FILE *fout = fopen((path+exp_name+".param").c_str(), "w");
        dnn->save(fout);
        fclose(fout);
    }
    //DMnistData<float> *test_data;// = new DMnistData<float>("../data", DData<float>::Test, 10000, false, dnn->handle());
    //test_data->set_devId(devId);
    //test_data = new DMnistData<float>("../data", DData<float>::Test, 10000, true, dnn->handle());
    //printf("Testing Error:%f\n", dnn->test(test_data));

    CUDA_CALL(hipDeviceReset());
#ifdef USE_MPI
    MPI_Finalize();
#endif
    return 0;
}
