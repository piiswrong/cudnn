#include "hip/hip_runtime.h"
#include <common.cuh>
#include <DNN.cuh>
#include <DData.cuh>

#include <nvml_old.h>
#include <sstream>

template<class T>
void fineTuneWithCheckpoint(DNN<T> *dnn, DData<T> *data, int ntotal, int ninterval, std::string path, int resuming) {
    for (int i = 0; i < ntotal; i += ninterval) {
        if (i + ninterval <= ntotal) 
            dnn->fineTune(data, ninterval);
        else 
            dnn->fineTune(data, ntotal - i);
        std::stringstream ss;
        ss << i+resuming;
        FILE *fout = fopen((path+"_"+ss.str()+".param").c_str(), "w");
        dnn->save(fout);
        fclose(fout);
    }
}

int main(int argc, char **argv) {
    std::string path = "/projects/grail/jxie/cudnn/log/";
    //feenableexcept(FE_DIVBYZERO | FE_INVALID | FE_OVERFLOW);
#ifdef USE_MPI 
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_world_rank);
#endif

    FILE *fin = NULL;
    char * exp_name = NULL;
    int resuming = -1;
    int devId = -1;
    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-') {
            switch (argv[i][1]) {
            case 'r': resuming = atoi(argv[i+1]); ++i; break;
            case 'd': devId = atoi(argv[i+1]); i++; break;
            default: printf("Invalid command line argument \'%c\'!\n", argv[i][1]); exit(-1); break;
            }
        }else {
            exp_name = argv[i];
        }
    }
    if (exp_name != NULL) {
        flog = fopen((path+exp_name+".log").c_str(), "w");
        fin = fopen((path+exp_name+".hyper").c_str(), "r");
        printf("Using configuration file %s\n", (path+exp_name+".hyper").c_str());
        if (fin == NULL) exit(-1);
    }

#ifndef DISABLE_GPU
    if (devId == -1) {
        nvmlReturn_t ret;
        unsigned int deviceCount;

        if ((ret = nvmlInit()) != NVML_SUCCESS)
        {
            printf("Could not init NVML: %s\n", nvmlErrorString(ret));
            return 1;
        }

        if ((ret = nvmlDeviceGetCount(&deviceCount)) != NVML_SUCCESS)
        {
            printf("Could not get device count: %s\n", nvmlErrorString(ret));
            nvmlShutdown();
            return 1;
        }
        printf("Device count: %d\n", deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            nvmlDevice_t device;
            if ((ret = nvmlDeviceGetHandleByIndex(i, &device)) != NVML_SUCCESS)
            {
                printf("Skip %d, can not get index\n", i);
                continue;
            }
            nvmlUtilization_t util;
            if ((ret = nvmlDeviceGetUtilizationRates(device, &util)) != NVML_SUCCESS)
            {
                printf("Can not get util rate on %d\n", i);
            }else {
                if (util.gpu < 5) 
                {
                    devId = i;
                    break;
                }
            }
        }
    }

    if (devId != -1)
    {
        printf("Selecting device %d\n", devId);
        CUDA_CALL(hipSetDevice(devId));
    }else {
        printf("Can not find idle device\n");
        exit(-1);
    }
#endif


    hipblasHandle_t handle = 0; 
    CUBLAS_CALL(hipblasCreate(&handle));

    int num_layers = 5;
    int hidden_dim = 799;
    char unit[255];
    strcpy(unit, "Oddroot");
    float pt_epochs = 0.0;
    DHyperParams _bp_hyper_params, _pt_hyper_params;
    _pt_hyper_params.idrop_out = false;
    _pt_hyper_params.idrop_rate = 0.5;
    _pt_hyper_params.hdrop_out = false;
    _pt_hyper_params.weight_decay = true;
    _pt_hyper_params.decay_rate = 0.01;
    _pt_hyper_params.momentum = 0.90;
    _pt_hyper_params.learning_rate = 0.01;

    _bp_hyper_params.check_interval = 10000;
    _bp_hyper_params.learning_rate = 0.1;
    _bp_hyper_params.idrop_out = false;
    _bp_hyper_params.idrop_rate = 0.2;
    _bp_hyper_params.hdrop_out = true;
    _bp_hyper_params.hdrop_rate= 0.2;
    _bp_hyper_params.momentum = 0.5;
    _bp_hyper_params.max_momentum = 0.90;
    _bp_hyper_params.step_momentum = 0.04;
    _bp_hyper_params.weight_decay = false;
    _bp_hyper_params.decay_rate = 0.000;
#ifdef ADMM
    _bp_hyper_params.decay_rate = 0.001;
#endif

    int bp_epochs = 200;
    if (fin != NULL) {
        READ_PARAM(num_layers);
        READ_PARAM(hidden_dim);
        fscanf(fin, "neuron=%s\n", unit);
        READ_PARAM(pt_epochs);
        READ_PARAM(bp_epochs);
        _pt_hyper_params.load(fin);
        _bp_hyper_params.load(fin);
    }


    int input_dim = 351, output_dim = 150;
    //input_dim = 28*28, output_dim = 10;
    int *layer_dims = new int[num_layers+1];
    layer_dims[0] = input_dim;
    layer_dims[num_layers] = output_dim;
    for (int i = 1; i < num_layers; i++) layer_dims[i] = hidden_dim;

    DNeuron<float> **neuron = new DNeuron<float>*[num_layers];
    for (int i = 0; i < num_layers-1; i++) {
        std::string str_unit(unit);
        if (str_unit == "Logistic") {
            neuron[i] = new DLogisticNeuron<float>(handle);
        }else if (str_unit == "Oddroot") {
            printf("using oddroot\n");
            neuron[i] = new DOddrootNeuron<float>(handle);
        }else if (str_unit == "ReLU") {
            neuron[i] = new DReLUNeuron<float>(handle);
        }else {
            printf("ERROR: \"%s\" is not a supported neuron type\n", unit);
            exit(-1);
        }
    }
    neuron[num_layers-1] = new DSoftmaxNeuron<float>(_bp_hyper_params.batch_size, handle);
    
    DNN<float> *dnn = new DNN<float>(num_layers, layer_dims, neuron, _pt_hyper_params, _bp_hyper_params, handle);
#ifdef ADMM
    DParallelMnistData<float> *data = new DParallelMnistData<float>("../data", mpi_world_size, mpi_world_rank, _bp_hyper_params.batch_size, dnn->handle());
    data->set_devId(devId);
    dnn->admmFineTune(data, 500);
#elif defined(DOWN_POUR_SGD)
    DParallelMnistData<float> *data = new DParallelMnistData<float>("../data", mpi_world_size - sgd_num_param_server, mpi_world_rank - sgd_num_param_server, _bp_hyper_params.batch_size, dnn->handle());
    data->set_devId(devId);
    dnn->fineTune(data, 500);

#else
    //DMnistData<float> *data = new DMnistData<float>("/scratch/jxie", DData<float>::Train, 50000, false, dnn->handle());
    //DData<float> *data = new DDummyData<float>(10,  handle);
    DTimitData<float> *data = new DTimitData<float>("/scratch/jxie/", 10000, false, dnn->handle());
#ifndef DISABLE_GPU
    data->set_devId(devId);
#endif
    if (resuming == -1 && pt_epochs > 0) dnn->pretrain(data, pt_epochs);
    if (resuming != -1) {
        printf("Resuming from %d-th epoch.\n", resuming);
        //std::stringstream ss;
        //ss << resuming - 10;
        fin = fopen((path+exp_name+".param").c_str(), "r");
        if (fin == 0) {
            printf("Error loading: cannot find file %s!\n", (path+exp_name+".param").c_str());
            exit(-1);
        }
        dnn->layers()[0]->weight()->samplePrint();
        dnn->load(fin);
        dnn->layers()[0]->weight()->samplePrint();
        fclose(fin);
        _bp_hyper_params.learning_rate *= std::pow(_bp_hyper_params.learning_rate_decay, resuming);
    }
    if (exp_name != NULL)
        fineTuneWithCheckpoint(dnn, data, bp_epochs, 10, path+exp_name, resuming);
    else 
        dnn->fineTune(data, bp_epochs);

#endif
    if (exp_name != NULL) {
        FILE *fout = fopen((path+exp_name+".param").c_str(), "w");
        dnn->save(fout);
        fclose(fout);
    }
    //DMnistData<float> *test_data;// = new DMnistData<float>("../data", DData<float>::Test, 10000, false, dnn->handle());
    //test_data->set_devId(devId);
    //test_data = new DMnistData<float>("../data", DData<float>::Test, 10000, true, dnn->handle());
    //printf("Testing Error:%f\n", dnn->test(test_data));

    CUDA_CALL(hipDeviceReset());
#ifdef USE_MPI
    MPI_Finalize();
#endif
    return 0;
}
