#include <common.cuh>
#include <DNN.cuh>
#include <DData.cuh>

int main() {
    hipblasHandle_t handle; 
    hipblasCreate(&handle);

    int num_layers = 2;
    int layer_dims[] = {28*28, 1023, 10};
    DHyperParams _bp_hyper_params, _pt_hyper_params;
    _bp_hyper_params.batch_size = 10;
    _bp_hyper_params.check_interval = 10;
    DNeuron<float> **neuron = new DNeuron<float>*[num_layers];
    neuron[0] = new DNeuron<float>(handle);
    neuron[1] = new DSoftmaxNeuron<float>(handle);
    
    DNN<float> *dnn = new DNN<float>(num_layers, layer_dims, neuron, _pt_hyper_params, _bp_hyper_params, handle);
    DMnistData<float> *data = new DMnistData<float>("../data", DData<float>::Train, _bp_hyper_params.batch_size, dnn->handle());
    //DData<float> *data = new DDummyData<float>(10,  handle);
    dnn->fineTune(data, 1);


    return 0;
}
