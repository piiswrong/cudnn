#include <common.cuh>
#include <DNN.cuh>
#include <DData.cuh>

int main(int argc, char **argv) {
#ifdef USE_MPI 
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_world_rank);
#endif
    hipblasHandle_t handle = 0; 
    CUDA_CALL(hipblasCreate(&handle));

    int num_layers = 2;
    int layer_dims[] = {28*28, 1024, 10};
    DHyperParams _bp_hyper_params, _pt_hyper_params;
    //_bp_hyper_params.batch_size = 10;
    _bp_hyper_params.check_interval = 10000;
    _bp_hyper_params.learning_rate = 1.0;
    _bp_hyper_params.idrop_out = false;
    _bp_hyper_params.hdrop_out = false;
    _bp_hyper_params.momentum = 0.0;
    _bp_hyper_params.max_momentum = 0.0;
#ifdef ADMM
    _bp_hyper_params.decay_rate = 0.01;
#endif

    //_bp_hyper_params.sparseInit = true;
    DNeuron<float> **neuron = new DNeuron<float>*[num_layers];
    neuron[0] = new DReLUNeuron<float>(handle);
    neuron[1] = new DSoftmaxNeuron<float>(_bp_hyper_params.batch_size, handle);
    
    DNN<float> *dnn = new DNN<float>(num_layers, layer_dims, neuron, _pt_hyper_params, _bp_hyper_params, handle);
#ifdef ADMM
    DParallelMnistData<float> *data = new DParallelMnistData<float>("../data", mpi_world_size, mpi_world_rank, _bp_hyper_params.batch_size, dnn->handle());
    dnn->admmFineTune(data, 50);
#else
    DMnistData<float> *data = new DMnistData<float>("../data", DData<float>::Train, 50000, false, dnn->handle());
    //DData<float> *data = new DDummyData<float>(10,  handle);
    dnn->fineTune(data, 10);
#endif

    DMnistData<float> *test_data;// = new DMnistData<float>("../data", DData<float>::Test, 10000, false, dnn->handle());
    test_data = new DMnistData<float>("../data", DData<float>::Test, 10000, true, dnn->handle());
    printf("Testing Error:%f\n", dnn->test(test_data));

    CUDA_CALL(hipDeviceReset());
#ifdef USE_MPI
    MPI_Finalize();
#endif
    return 0;
}
