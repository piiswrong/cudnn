#include "hip/hip_runtime.h"
#include <kernels.cuh>
#include <common.cuh>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void kSetupCurand(hiprandState *state, int nelem, unsigned int seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nelem) hiprand_init(seed, i, 0, &state[i]);
}

